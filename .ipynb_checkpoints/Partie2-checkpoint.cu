//Partie 2

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <cstdlib>


//------------------------------ Macros ---------------------------------------------------------------------

void MatrixInitRand(float *M, int n);
void MatrixInitZero(float *M, int n);
void MatrixPrint(float *M, int n);

__global__ void cudaConv(float *In, float *Kernel, float *Out, int Nx, int Ny, int kernel_size);
__global__ void cudaMoyen2(float *E, float *F, int n);

// 3.1


// ----------------------------- Fontions utilitaires ------------------------------------------------------------------------

void MatrixInitRand(float *M, int n){
    for (int i = 0; i < n; i++){
        M[i] = (float)(rand()%1000)/1000 ; 
        //flottant entre 0 et 1 de précision 10⁻3
    }
}

void MatrixInitZero(float *M, int n){
    for (int i = 0; i < n; i++){
        M[i] = 0 ; 
        //flottant entre 0 et 1 de précision 10⁻3
    }
}

//nb_mat est le nombre de matrices
//n c'est la taille de la matrice n*n
void MatrixPrint(float *M, int n,int nb_mat){
    for (int i = 0; i < n*n*nb_mat ; i++){
        if((i+1)%n ==0){
            printf("%1.5f\n",M[i]);
            if((i+1)%n*n ==0){printf("\n");}
        }else{
            printf("%1.5f ",M[i]);
        }
        
    }
    printf("\n");
}

// 3.2

//--------------------------------- CudaConv et CudaMoyen2 -----------------------------------------------------------------------------

__global__ void cudaConv(float *In, float *Kernel, float *Out, int Nx, int Ny, int kernel_size){
    
    
    //Nx and Ny are the dimensions of both the original and convoluted image, and kernel_size is the dimension of the convolution kernel.
    
    //each block is assigned to a row of an image, iy integer index of y
    int iy = blockIdx.x + (kernel_size - 1)/2;
    
    //each thread is assigned to a pixel of a row, ix integer index of x
    int ix = threadIdx.x + (kernel_size - 1)/2;
    
    //center of kernel in both dimensions
    int center = (kernel_size -1)/2;
    
    //For each block thread, the memory location of the corresponding pixel can be calculated by:
    int idx = iy*Nx +ix;
    int ki;int ii;int jj;
    
 
    int tid = threadIdx.x;
    int K2 = kernel_size*kernel_size;
    extern __shared__ float sdata[]; //we store the kernel in shared memory
    if (tid<K2){
        sdata[tid] = Kernel[tid];
        __syncthreads();
    }
        
    if (idx<Nx*Ny){
        int sum =0;
        for (ki = 0; ki<kernel_size; ki++){
            for (int kj = 0; kj<kernel_size; kj++){
                ii = kj + ix - center;
                jj = ki + iy - center;
                sum+=In[jj*Nx+ii]*sdata[ki*kernel_size + kj];
            }
        }
        Out[idx] = sum;
    }
    
}

__global__ void cudaMoyen2(float *E, float *S, int n){
    // n = taille d'une ligne de E (et aussi d'une colonne)
    
    //1er élément du 1er dim3 = nombre matrices 2D de E
    int nb_mat = blockIdx.x;
    //nb_mat * taille d'une matrice de S (= taille du shift dans l'indice de S):
    int shift_S = nb_mat * n/2 * n/2 ;
    //nb_mat * taille d'une matrice de E (= taille du shift dans l'indice de E):
    int shift_E = nb_mat * n * n ;
    
    //2e élément du 1er dim3 = nombre de colonnes/2 de E = nombre de col de S
    int output_col = blockIdx.y; 
    //2e dim3 (contient 1 seul élément) = nombre de lignes/2 de E = nombre de lignes de S
    int output_row = threadIdx.x;
    
    //on se déplace de 2 en 2 dans les matrices d'entrée
    int input_col = 2 * output_col;
    int input_row = 2 * output_row;
    
    //Calcul de S en fonction de E :
    S[shift_S + output_row * n + output_col] = (float)(( E[shift_E + input_row * n + input_col] + E[shift_E + (input_row+1) * n + input_col] + E[shift_E + input_row * n + (input_col+1)] + E[shift_E + (input_row+1) * n + (input_col+1)] )/4);
}


// ----------------------------------------------- Main -----------------------------------------------------------------------------

int main(){
    
    // 3.1 
    
    // ----------------------------- Initialisation des tailles -----------------------------
    
    //matrice raw_data
    int n1 = 32; //size of input image
    const int ARRAY_SIZE1 = n1*n1;
    const int ARRAY_BYTES1 = ARRAY_SIZE1 * sizeof(float);
    
    //matrice C1_data
    int n21 = 28; // size of output image of conv1
    int n22 = 6; // nb of features maps in output of conv1
    const int ARRAY_SIZE2 = n21*n21*n22;
    const int ARRAY_BYTES2 = ARRAY_SIZE2 * sizeof(float);
    
    //matrice S1_data : issue du sous-échantillonnage de facteur 2 de C1_data
    int n31 = 14; //size of output image after S1
    int n32 = 6; // nb of feature maps
    const int ARRAY_SIZE3 = n31*n31*n32;
    const int ARRAY_BYTES3 = ARRAY_SIZE3* sizeof(float);
    
    //matrice C1_kernel : 6 noyaux de conv de taille 5x5
    int kernel_size = 5;
    int nb_of_kernels = 6;
    const int ARRAY_SIZE4 = kernel_size*kernel_size*nb_of_kernels;
    const int ARRAY_BYTES4 = ARRAY_SIZE4 * sizeof(float);
    
    //allocation de mémoire pour les matrices sur CPU
    float *raw_data, *C1_data, *S1_data, *C1_kernel;
    raw_data = (float*)malloc(ARRAY_BYTES1);
    C1_data = (float*)malloc(ARRAY_BYTES2);
    S1_data = (float*)malloc(ARRAY_BYTES3);
    C1_kernel = (float*)malloc(ARRAY_BYTES4);
    
     
    //------------------------------------------Initialisation des matrices ----------------------------------------------------------------
    
    MatrixInitRand(raw_data, ARRAY_SIZE1);
    MatrixInitZero(C1_data, ARRAY_SIZE2);
    MatrixInitZero(S1_data, ARRAY_SIZE3);
    MatrixInitRand(C1_kernel, ARRAY_SIZE4);
    
    // pour tester :
    //MatrixPrint(C1_data, n21* n21* n22);
    
    
    // 3.2
       
    //-------------------------------------- Allocation de mémoire sur GPU --------------------------------------------------------------------
    
    float *d_raw_data, *d_C1_data, *d_S1_data, *d_C1_kernel;
    hipMalloc((void **) &d_raw_data, ARRAY_BYTES1);
    hipMalloc((void **) &d_C1_data, ARRAY_BYTES2);
    hipMalloc((void **) &d_S1_data, ARRAY_BYTES3);
    hipMalloc((void **) &d_C1_kernel, ARRAY_BYTES4);
    
    
    //transfert de données pour le calcul sur gpu
    //entrée:
    hipMemcpy(d_raw_data, raw_data, ARRAY_BYTES1, hipMemcpyHostToDevice);
    //sortie:
    hipMemcpy(d_C1_data, C1_data, ARRAY_BYTES2, hipMemcpyHostToDevice);
    //filtre:
    hipMemcpy(d_C1_kernel, C1_kernel, ARRAY_BYTES4, hipMemcpyHostToDevice);
    
    
    // ------------------------------------------------Layer 2 : convolution ---------------------------------------------------------
    dim3 my_blocks_conv (n21, n21, 1); // 1 psk dim3
    cudaConv<<<my_blocks_conv, ARRAY_BYTES4>>>(d_raw_data,d_C1_kernel, d_C1_data, n1,n21,kernel_size);
    
    hipMemcpy(C1_data, d_C1_data, ARRAY_BYTES2, hipMemcpyDeviceToHost);
    MatrixPrint(C1_data,n21,1);
    
    // ----------------------------------------------- Layer 3 : moyenneur ---------------------------------------
    
    dim3 my_blocks (n32, n31, 1); // taille = 6 * 28, on préfère regrouper comme ça
    //plutôt que 28*28 qui sera + gros 
    cudaMoyen2<<< my_blocks, n31>>>(d_C1_data,d_S1_data, n31);
    //ici, n32 = blockId.x et n31 = blockId.y pour se repérer dans la fonction
    hipMemcpy(S1_data, d_S1_data, ARRAY_BYTES3, hipMemcpyDeviceToHost);
    MatrixPrint(S1_data,n31,1);
    
    //---------------------------------------------------- Libération des ressources -------------------------------------- 
    hipFree(d_raw_data);
    hipFree(d_C1_data);
    hipFree(d_S1_data);
    hipFree(d_C1_kernel);
    
    free(raw_data);
    free(C1_data);
    free(S1_data);
    free(C1_kernel);
    
    // This call waits for all of the submitted GPU work to complete
    hipDeviceSynchronize();
    
    return 0;
}