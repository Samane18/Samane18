#include "hip/hip_runtime.h"
/******************************************* PARTIE 2 ************************************************/

/* L'objectif de cette deuxième partie est d'implémenter un petit réseau convolutionnel qui va 
*  convoluer une image d'entrée raw_data de taille 32*32 par un une série de 6 kernels de taille 5*5
*  stockés dans la matrice C1_kernels. Nous obtenons en sortie de cette couche 6 feature maps 
*  de taille 28*28 (la taille 28 résultant de l'opération (32 - 5 + 1)), qui nous donnent la matrice 
*  C1_data. La deuxième couche est une étape de sous-échantillonage par 2 qui va, sur chaque feature map,
*  moyenner chaque carré de pixels de taille de 2*2. Cela nous donne 6 matrices de taille 14*14, la 
*  taille des feature maps étant divisée par deux, et nous appliquons à chaque pixels de ces matrices
*  la fonction d'activation tanh (qui ramène les valeurs entre -1 et 1) et nous stockons les 6 matrices
*  14*14 dans la matrice S1_data.
*
*  Nous allons effectuer plusieurs tests pour montrer que notre code marche bien, en montrant les 
*  résultats d'abord sur des matrices simples de petite taille pour raw_data et C1_data, dont nous 
*  connaissons le résultat de la convolution et du sous-échantillonnage. Nous montrerons en dernier le 
*  fonctionnement du code avec les matrices raw_data et C1_kernel initialisées avec des nombres 
*  aléatoires entre 0 et 1.
*  
*  Le code comporte d'abord les fonctions que nous utilisons pour les différentes initialisations des 
*  matrices, puis les codes des fonctions de convolution, de sous-échantillonnage et de tanh. Nous avons
*  ensuite le main, dont on expliquera la structure plus bas dans le code, juste avant son début.
*/

// ----------------------------------------- Includes ------------------------------------------------
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#define BLOCK_SIZE 32


// ------------------------------- Fonctions d'initialisation de matrices ----------------------------

/* Ces premières fonctions nous aident à intialiser les matrices raw_data, C1_data, C1_kernel, et 
*  S1_data. Chaque fonction prend en argument 
*
*/

/*
 * Fonction: MatrixInitInt
 * ----------------------------
 *   Initialise chaque valeur M[i] d'un tableau M par la valeur i 
 *   M  : tableau à initialiser
 *   size : nombre total d'éléments dans M
 */
void MatrixInitInt(float *M,int size)
{
    for(int i=0;i<size;i++){
        M[i]=i;
    }
}

/*
 * Fonction: MatrixInitOne
 * ----------------------------
 *   Initialise chaque valeur M[i] d'un tableau M par la valeur 1
 *   M  : tableau à initialiser
 *   size : nombre total d'éléments dans M
 */

void MatrixInitOne(float *M,int size)
{
    for(int i=0;i<size;i++){
        M[i]=1;
    }
}

/*
 * Fonction: MatrixInitZero
 * ----------------------------
 *   Initialise chaque valeur M[i] d'un tableau M par la valeur 0 
 *   M  : tableau à initialiser
 *   size : nombre total d'éléments dans M
 */

void MatrixInitZero(float *M,int size)
{
    for(int i=0;i<size;i++){
        M[i]=0;
    }
}

void MatrixInitRand(float *M, int size){
    for (int i = 0; i<size; i++){
        M[i] = (float)(rand()%1000)/1000 ; 
        //flottant entre 0 et 1 de précision 10⁻3
    }
}


void MatrixPrint(float *C,const int nx,const int ny,const int nz)
{
    float *ic=C;
    printf("\n Matrix: (%d*%d*%d) \n",nx,ny,nz);
    for(int k=0;k<nz;k++){
        for(int i=0;i<ny;i++){
            for(int j=0;j<nx;j++){
                if(ic[k*(nx*ny)+nx*i +j]<0){
                    printf("%3.1f ",ic[k*(nx*ny)+nx*i +j]);                     
                }else{
                    printf(" %3.1f ",ic[k*(nx*ny)+nx*i +j]);
                }
            }
            printf("\n");

        }
        printf("\n");
    }
}

// Fonction qui sert à afficher l'indexage globale d'un thread sur une grille, pour mieux comprendre l'indexage 
// A servi simplement de helper function
__global__ void printthreadindex(float *A,const int nx,const int ny)
{
    int ix=threadIdx.x+blockIdx.x*blockDim.x;
    int iy=threadIdx.y+blockIdx.y*blockDim.y;
    //int iz=threadIdx.z+blockIdx.z*blockDim.z;
    
    
    unsigned int idx=ix+iy*nx; //+nx*ny*iz;

    printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) global index %2d  ival %2d \n",threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,ix,iy,idx,A[idx]);

}


// --------------------------------- Convolution, sous-échantillonnage, et fonction d'activation ------------------------------------

// Fonction exécutée par le device (GPU) et appelée par le device (GPU).
// Elle doit être appellée dans un kernel et ne nécessite d'appel <<<B,T>>> comme les fonctions __global__
// On l'appelle à la fin de la fonction cudaMoyen2 
__device__ float activation_tanh(float M){
    return tanhf(M);
}


// Fonction qui réaliser la convolution d'une matrice avec 1 SEUL kernel, donne en sortie 1 seul feature map
__global__ void gpuMatrix2DConv(float* gpuMat1, float* kernel, float* gpuMat3, int m1Rows, int m1Cols, int mRowsCols, int m3Rows, int m3Cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0;

    if (row < m3Rows && col < m3Cols) {
        for (int maskRow = 0; maskRow < mRowsCols; maskRow++) {
            for (int maskCol = 0; maskCol < mRowsCols; maskCol++) {
                sum += gpuMat1[(row + maskRow) * m1Cols + (col + maskCol)] * kernel[maskRow * mRowsCols + maskCol];
            }
        }
        gpuMat3[row * m3Cols + col] = sum;
    }
}

// On part de la fonction de convolution 2D pour construire la convolution 3D : à chaque thread, on calcule la valeur du pixel (i,j) de CHAQUE feature map. Un thread réalise donc nb_features_maps (ici 6) convolutions avec un carré de l'image
__global__ void gpuMatrix3DConv(float* gpuMat1, float* kernel, float* gpuMat3, int m1Rows, int m1Cols, int kernel_size, int nb_kernels, int m3Rows, int m3Cols){
    
    //Identifiants globaux ligne et colonne de la matrice gpuMat1
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < m3Rows && col < m3Cols) {
        
        // On itère l'opération de convolution d'un carré kernel_size*kernel_size de la matrice d'entrée sur chacun des nb_kernels kernels 
        for(int num_kernel = 0; num_kernel<nb_kernels; num_kernel++){
            // L'offset ici correpond au numéro de kernel dans avec lequel on fait une opération
            int offset = num_kernel*kernel_size*kernel_size;
            //On initialise la somme qui donnera la valeur final du pixel (row,col) de la matrice C1_data
            float sum = 0.0;
            for (int maskRow = 0; maskRow < kernel_size; maskRow++) {
                for (int maskCol = 0; maskCol < kernel_size; maskCol++) {
                    sum += gpuMat1[(row + maskRow) * m1Cols + (col + maskCol) ] * kernel[maskRow * kernel_size + maskCol + offset];
                }
            }
            gpuMat3[num_kernel*(m3Rows*m3Cols) + row * m3Cols + col] = sum;
        
        }
    }
    
}

/* fonction moyenneur executée sur GPU 
L'argument de la fonction correspond à la dimension de la matrice d'entrée
Les nombres de blocks et threads sont ceux de la matrice d'arrivés car le nombre de calculs corespond au nombre d'éléments à l'arrivée
n : taille de la matrice d'entrée
*/
__global__ void cudaMoyen2(float *E, float *S, int n){
    // n = taille d'une ligne de E (et aussi d'une colonne)
    
    int n_out = n/2; // dimension de la matrice de sortie
    
    //1er élément du 1er dim3 = nombre matrices 2D de E
    int nb_mat = blockIdx.x;
    
    //nb_mat * taille d'une matrice de S (= taille du shift dans l'indice de S):
    int shift_S = nb_mat * n_out * n_out ;
    //nb_mat * taille d'une matrice de E (= taille du shift dans l'indice de E):
    int shift_E = nb_mat * n * n ;
    
    //2e élément du 1er dim3 = nombre de colonnes/2 de E = nombre de col de S
    int output_col = blockIdx.y; 
    
    //2e dim3 (contient 1 seul élément) = nombre de lignes/2 de E =  nombre de lignes de S
    int output_row = threadIdx.x;
    
    //on se déplace de 2 en 2 dans les matrices d'entrée
    int input_col = 2 * output_col;
    int input_row = 2 * output_row;
    
    //Calcul pour chaque élément de S la moyenne en fonction des éléments de E :
    S[shift_S + output_row * n_out + output_col] = (float)(( E[shift_E + input_row * n + input_col] + E[shift_E + (input_row+1) * n + input_col] + E[shift_E + input_row * n + (input_col+1)] + E[shift_E + (input_row+1) * n + (input_col+1)] )/4);
    
}



int main()
{
    //Matrix raw_data
    int raw_size=8;
    float rawBytes=raw_size*raw_size*sizeof(float);

    float *raw_data;
    raw_data=(float *)malloc(rawBytes);

    MatrixInitOne(raw_data,raw_size*raw_size);
    MatrixPrint(raw_data,raw_size,raw_size,1);
    
    float *d_raw_data;
    hipMalloc((void **)&d_raw_data,rawBytes);

    hipMemcpy(d_raw_data,raw_data,rawBytes,hipMemcpyHostToDevice);
    
    
    //Matrix C1_Kernel
    int C1_kernel_size=3,nb_kernels=6;
    float C1_kernelBytes=C1_kernel_size*C1_kernel_size*nb_kernels*sizeof(float);
    
    float *C1_kernel;
    C1_kernel=(float *)malloc(C1_kernelBytes);
    
    MatrixInitInt(C1_kernel,C1_kernel_size*C1_kernel_size*nb_kernels);
    MatrixPrint(C1_kernel,C1_kernel_size,C1_kernel_size,nb_kernels);
    
    float *d_C1_kernel;
    hipMalloc((void **)&d_C1_kernel,C1_kernelBytes);

    hipMemcpy(d_C1_kernel,C1_kernel,C1_kernelBytes,hipMemcpyHostToDevice);
    
    
    //Matrix C1_data output of convolution 1
    int C1_data_size=6,nb_of_maps=6;
    float C1_data_Bytes=C1_data_size*C1_data_size*nb_of_maps*sizeof(float);
    
    float *C1_data;
    C1_data=(float *)malloc(C1_data_Bytes);

    float *d_C1_data;
    hipMalloc((void **)&d_C1_data,C1_data_Bytes);

    MatrixInitZero(C1_data,C1_data_size*C1_data_size*nb_of_maps);
    MatrixPrint(C1_data,C1_data_size,C1_data_size,nb_of_maps);
    
    hipMemcpy(d_C1_data,C1_data,C1_data_Bytes,hipMemcpyHostToDevice);
    
    //Matrix S1_data output of ss-ech 1
    int S1_data_size=3;
    float S1_data_Bytes=S1_data_size*S1_data_size*nb_of_maps*sizeof(float);
    
    float *S1_data;
    S1_data=(float *)malloc(S1_data_Bytes);

    float *d_S1_data;
    hipMalloc((void **)&d_S1_data,S1_data_Bytes);

    MatrixInitZero(S1_data,S1_data_size*S1_data_size*nb_of_maps);
    MatrixPrint(S1_data,S1_data_size,S1_data_size,nb_of_maps);
    
    hipMemcpy(d_S1_data,S1_data,S1_data_Bytes,hipMemcpyHostToDevice);
    
    // Process
    //dim3 block(raw_size);//,3);
    //dim3 grid(raw_size);
    //printthreadindex <<<grid,block>>> (d_MatA,nx,ny);//,1);
    
    //printthreadindex <<<grid,block>>> (d_raw_data,raw_size,raw_size);//,1);

    
    // --------------- Convolution 1 -----------------------------
    int threadsPerBlock = 32;
    int gridCols = ceil(double(C1_data_size) / double(threadsPerBlock));
    int gridRows = ceil(double(C1_data_size) / double(threadsPerBlock));

    dim3 gridDim(gridCols, gridRows);
    dim3 blockDim(threadsPerBlock, threadsPerBlock);	// total 32x32=1024 threads
    //gpuMatrix2DConv << < gridDim, blockDim >> > (d_raw_data, d_C1_kernel, d_C1_data, raw_size, raw_size, C1_kernel_size, C1_data_size, C1_data_size);
    gpuMatrix3DConv << < gridDim, blockDim >> > (d_raw_data, d_C1_kernel, d_C1_data, raw_size, raw_size, C1_kernel_size,nb_kernels, C1_data_size, C1_data_size);
    
    // ------------------ Sous-échantillonage 1 ------------------------------
    dim3 my_blocks (nb_of_maps, S1_data_size, 1);
    cudaMoyen2<<<my_blocks,S1_data_size>>>(d_C1_data,d_S1_data, C1_data_size);
    
    
    
    // Retour au CPU
    hipMemcpy(C1_data, d_C1_data, C1_data_Bytes, hipMemcpyDeviceToHost); // C1_data
    hipMemcpy(S1_data, d_S1_data, S1_data_Bytes, hipMemcpyDeviceToHost); // C1_data
    
    // Affichage Conv et Sous-échantillonage 
    printf("\nConvolution\n");
    MatrixPrint(C1_data,C1_data_size,C1_data_size,nb_of_maps);

    printf("\nSous-échantillonage\n");
    MatrixPrint(S1_data,S1_data_size,S1_data_size,nb_of_maps);

    // Libération des ressources 
    hipFree(d_raw_data);
    hipFree(d_C1_kernel);
    hipFree(d_C1_data);
    
    free(raw_data);
    free(C1_kernel);
    free(C1_data);
    
    // This call waits for all of the submitted GPU work to complete
    hipDeviceSynchronize();
    
    return 0;

}