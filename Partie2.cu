//Partie 2

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <cstdlib>
#define BLOCK_SIZE 32

//------------------------------ Macros ---------------------------------------------------------------------

void MatrixInitRand(float *M, int n);
void MatrixInitZero(float *M, int n);
void MatrixPrint(float *M, int n, int p);

__global__ void cudaConv(float *In, float *Kernel, float *Out, int Nin, int Nout, int kernel_size);
__global__ void cudaMoyen2(float *E, float *F, int n);
__global__ void convolution_2d(float *matrix, float* kernel, float *result, int N, int k);
// 3.1


// ----------------------------- Fontions utilitaires ------------------------------------------------------------------------

void MatrixInitRand(float *M, int n){
    for (int i = 0; i < n; i++){
        M[i] = (float)(rand()%1000)/1000 ; 
        //flottant entre 0 et 1 de précision 10⁻3
    }
}

void MatrixInitZero(float *M, int n){
    for (int i = 0; i < n; i++){
        M[i] = 0 ; 
        //flottant entre 0 et 1 de précision 10⁻3
    }
}


//nb_mat est le nombre de matrices
//n c'est la taille de la matrice n*n
//void MatrixPrint(float *M, int n,int nb_mat){
//    printf("\n");
//    for (int i = 0; i < n*n*nb_mat ; i++){
//        if((i+1)%n ==0){
//            printf("%1.5f\n",M[i]);
//            if((i+1)%n*n ==0){printf("\n");}
//        }else{
//            printf("%1.5f ",M[i]);
//        }
        
//    }
//}

void MatrixPrint(float *M, int n, int p){
    
    int i,j;
    
    for(i=0; i<n; i++) {
          for(j=0;j<p;j++) {
              if(M[i*p +j]>0) printf(" ");
             printf("%1.2f ", M[i*p +j]);
             if(j==n-1){
                printf("\n");
             }
          }
    }
    printf("\n");
}


// 3.2

//--------------------------------- CudaConv et CudaMoyen2 -----------------------------------------------------------------------------

__global__ void cudaConv(float *In, float *Kernel, float *Out, int Nin, int Nout, int kernel_size){
    
    
    //Nin and Nout are the dimensions of both the original and convoluted image, and kernel_size is the dimension of the convolution kernel.
    
    //each block is assigned to a row of an image, iy integer index of y
    int iy = blockIdx.x + (kernel_size - 1)/2;
    
    //each thread is assigned to a pixel of a row, ix integer index of x
    int ix = threadIdx.x + (kernel_size - 1)/2;
    
    //center of kernel in both dimensions, kernel_size impair  
    int center = (kernel_size -1)/2;  // ici (5-1)/2 = 2
    
    //For each block thread, the memory location of the corresponding pixel can be calculated by:
    int idx = iy*Nin +ix;
    int ki;int ii;int jj;
    
 
    int tid = threadIdx.x;
    int K2 = kernel_size*kernel_size;
    extern __shared__ float sdata[]; //we store the kernel in shared memory
    if (tid<K2){
        sdata[tid] = Kernel[tid];
        __syncthreads();
    }
        
    if (idx<Nin*Nout){
        int sum =0;
        for (ki = 0; ki<kernel_size; ki++){
            for (int kj = 0; kj<kernel_size; kj++){
                ii = kj + ix - center;
                jj = ki + iy - center;
                sum+=In[jj*Nin+ii]*sdata[ki*kernel_size + kj];
            }
        }
        Out[idx] = sum;
    }
    
}

__global__ void cudaMatrixConv(float *M, float *K, float *Mout, int n, int k)
{
    int nbmatrix = gridDim.x;
    int m = blockIdx.x;
    int l = threadIdx.x;
    int c = threadIdx.y;
    int el = m*blockDim.x*blockDim.y + l*blockDim.y + c; //element d'indice (x,y)
    
    //printf("nbmatrix : %d \n",nbmatrix);
    //printf("m : %d \n",m);
    //printf("blockDim.x : %d \n",blockDim.x);
    //printf("blockDim.y : %d \n",blockDim.y );
    int center = k/2;
    int originsize = blockDim.x+2*center;
    //printf("originsize : %d \n",originsize );
    // Handling arbitrary vector size
    if (el < nbmatrix*n*n){
        float sum = 0;
        
        for(int kc=0; kc<k; kc++){
            for(int kl=0; kl<k; kl++){
                int kel = m*k*k + kl*k + kc;
                int Mel = (l + kl)*originsize + (c + kc);
                sum = sum + M[Mel] * K[kel];
            }
        }
        Mout[el] = sum;
    }
}

// 2D Convolution Kernel
// Takes:
//  matrix: Input matrix
//  result: Convolution result
//  N:      Dimensions of the matrix
//  k:      Dimensions of the kernel

__global__ void convolution_2d(float *matrix, float* kernel, float *result, int N, int k) {
    
    // Calculate the global thread positions
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    int MASK_OFFSET = k/2; //Kernel size / 2

    // Starting index for calculation
    int start_r = row; 
    int start_c = col;
    
    // Temp value for accumulating the result
    float temp = 0;

    // Iterate over all the rows
    for (int i = 0; i < k; i++) {
        // Go over each column
        for (int j = 0; j < k; j++) {
            // Range check for rows
            if ((start_r + i) >= 0 && (start_r + i) < N) {
                // Range check for columns
                if ((start_c + j) >= 0 && (start_c + j) < N) {
                    // Accumulate result
                    temp += matrix[(start_r + i) * N + (start_c + j)] * kernel[i * k + j];
                }
            }
        }
    }

    // Write back the result
    result[row * N + col] = temp;
    
}


__global__ void cudaMoyen2(float *E, float *S, int n){
    // n = taille d'une ligne de E (et aussi d'une colonne)
    
    //1er élément du 1er dim3 = nombre matrices 2D de E
    int nb_mat = blockIdx.x;
    //nb_mat * taille d'une matrice de S (= taille du shift dans l'indice de S):
    int shift_S = nb_mat * n/2 * n/2 ;
    //nb_mat * taille d'une matrice de E (= taille du shift dans l'indice de E):
    int shift_E = nb_mat * n * n ;
    
    //2e élément du 1er dim3 = nombre de colonnes/2 de E = nombre de col de S
    int output_col = blockIdx.y; 
    //2e dim3 (contient 1 seul élément) = nombre de lignes/2 de E = nombre de lignes de S
    int output_row = threadIdx.x;
    
    //on se déplace de 2 en 2 dans les matrices d'entrée
    int input_col = 2 * output_col;
    int input_row = 2 * output_row;
    
    //Calcul de S en fonction de E :
    S[shift_S + output_row * n + output_col] = (float)(( E[shift_E + input_row * n + input_col] + E[shift_E + (input_row+1) * n + input_col] + E[shift_E + input_row * n + (input_col+1)] + E[shift_E + (input_row+1) * n + (input_col+1)] )/4);
}


// ----------------------------------------------- Main -----------------------------------------------------------------------------

int main(){
    
    // 3.1 
    
    // ----------------------------- Initialisation des tailles -----------------------------
    
    //matrice raw_data
    int n1 = 32; //size of input image
    const int ARRAY_SIZE1 = n1*n1;
    const int ARRAY_BYTES1 = ARRAY_SIZE1 * sizeof(float);
    
    //matrice C1_data
    int n21 = 28; // size of output image of conv1
    int n22 = 1; // nb of features maps in output of conv1
    const int ARRAY_SIZE2 = n21*n21*n22;
    const int ARRAY_BYTES2 = ARRAY_SIZE2 * sizeof(float);
    
    //matrice S1_data : issue du sous-échantillonnage de facteur 2 de C1_data
    int n31 = 14; //size of output image after S1
    int n32 = 1; // nb of feature maps
    const int ARRAY_SIZE3 = n31*n31*n32;
    const int ARRAY_BYTES3 = ARRAY_SIZE3* sizeof(float);
    
    //matrice C1_kernel : 6 noyaux de conv de taille 5x5
    int kernel_size = 5;
    int nb_of_kernels = 1;
    const int ARRAY_SIZE4 = kernel_size*kernel_size*nb_of_kernels;
    const int ARRAY_BYTES4 = ARRAY_SIZE4 * sizeof(float);
    
    //allocation de mémoire pour les matrices sur CPU
    float *raw_data, *C1_data, *S1_data, *C1_kernel;
    raw_data = (float*)malloc(ARRAY_BYTES1);
    C1_data = (float*)malloc(ARRAY_BYTES2);
    S1_data = (float*)malloc(ARRAY_BYTES3);
    C1_kernel = (float*)malloc(ARRAY_BYTES4);
    
     
    //------------------------------------------Initialisation des matrices ----------------------------------------------------------------
    
    MatrixInitRand(raw_data, ARRAY_SIZE1);
    MatrixInitZero(C1_data, ARRAY_SIZE2);
    MatrixInitZero(S1_data, ARRAY_SIZE3);
    MatrixInitRand(C1_kernel, ARRAY_SIZE4);
    
    // pour tester :
    printf("\nMatrice de données M\n");
    MatrixPrint(raw_data, n1,n1);
    printf("\nKernel\n");
    MatrixPrint(C1_kernel, kernel_size,kernel_size);   
    printf("\nMatrice C1_data avant convolution\n");
    MatrixPrint(C1_data, n21,n21);   
    
    
    // 3.2
       
    //-------------------------------------- Allocation de mémoire sur GPU --------------------------------------------------------------------
    
    float *d_raw_data, *d_C1_data, *d_S1_data, *d_C1_kernel;
    hipMalloc((void **) &d_raw_data, ARRAY_BYTES1);
    hipMalloc((void **) &d_C1_data, ARRAY_BYTES2);
    hipMalloc((void **) &d_S1_data, ARRAY_BYTES3);
    hipMalloc((void **) &d_C1_kernel, ARRAY_BYTES4);
    
    
    //transfert de données pour le calcul sur gpu
    //entrée:
    hipMemcpy(d_raw_data, raw_data, ARRAY_BYTES1, hipMemcpyHostToDevice);
    //sortie:
    hipMemcpy(d_C1_data, C1_data, ARRAY_BYTES2, hipMemcpyHostToDevice);
    //filtre:
    hipMemcpy(d_C1_kernel, C1_kernel, ARRAY_BYTES4, hipMemcpyHostToDevice);
    
    
    // ------------------------------------------------Layer 2 : convolution ---------------------------------------------------------
    //int max_threads = 1024;
    //dim3 nbThreads (n1, n1, 1); // 1 psk dim3
    //dim3 nbBlocks  (n1*n1/1024);
    
    // Calculate grid dimensions
    int THREADS = 32; //32*32=1024 threads max par bloc
    int BLOCKS = (n1+THREADS-1) / THREADS; //= 512/32=16 => 16*16=256 blocs

    // Dimension launch arguments
    dim3 block_dim(THREADS,THREADS);
    dim3 grid_dim(BLOCKS,BLOCKS);

    // Perform 2D Convolution
    convolution_2d<<<32,32>>>(d_raw_data,d_C1_kernel, d_C1_data, n21 ,kernel_size);
    hipMemcpy(C1_data, d_C1_data, ARRAY_BYTES2, hipMemcpyDeviceToHost);
    
    printf("\nConvolution\n");
    MatrixPrint(C1_data,28,28);
    
    // ----------------------------------------------- Layer 3 : moyenneur ---------------------------------------
    
    dim3 my_blocks (n32, n31, 1); // taille = 6 * 28, on préfère regrouper comme ça
    //plutôt que 28*28 qui sera + gros 
    cudaMoyen2<<< my_blocks, n31>>>(d_C1_data,d_S1_data, n31);
    //ici, n32 = blockId.x et n31 = blockId.y pour se repérer dans la fonction
    hipMemcpy(S1_data, d_S1_data, ARRAY_BYTES3, hipMemcpyDeviceToHost);
    //MatrixPrint(S1_data,n31,1);
    
    //---------------------------------------------------- Libération des ressources -------------------------------------- 
    hipFree(d_raw_data);
    hipFree(d_C1_data);
    hipFree(d_S1_data);
    hipFree(d_C1_kernel);
    
    free(raw_data);
    free(C1_data);
    free(S1_data);
    free(C1_kernel);
    
    // This call waits for all of the submitted GPU work to complete
    hipDeviceSynchronize();
    
    return 0;
}

